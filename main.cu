#include "hip/hip_runtime.h"
#pragma  once
#include "hip/hip_runtime.h"
#include ""
#include <cv.h>
#include <highgui.h>
#include <stdio.h>

using namespace std;
using namespace cv;

__global__ void Edge(uchar * inMAP, uchar * outMAP) {

    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    int Dim = gridDim.x *blockDim.x;
    int offset = x + y *Dim;
    
	if (inMAP[offset] < 127)
	{
	    outMAP[offset] =0;
	}
	else
	{
	    outMAP[offset] =255;
	}
}

extern "C" int run(IplImage *frame, PCA *decPCA, Mat pRecon, int threshold)
{
	uchar     *inMAP;
	uchar     *outMAP;
	uchar     *data = (uchar*)frame->imageData;
	///////////////////////////////////////////
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    assert(cudaStatus == hipSuccess);
	
	cudaStatus = hipMalloc((void**)&inMAP, frame->height*frame->width* sizeof(uchar));
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&outMAP, frame->height*frame->width* sizeof(uchar));
    assert(cudaStatus == hipSuccess);
	/////////////////////////////////////////////
	int size = frame->height* frame->width* sizeof(uchar);
    cudaStatus = hipMemcpy(inMAP, data, size, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

	dim3 grid(frame->width,frame->height);
    dim3 block(1,1);

	Edge<<<grid, block>>>(inMAP, outMAP);
	cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMemcpy(data, outMAP, size, hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

	return 0;
}