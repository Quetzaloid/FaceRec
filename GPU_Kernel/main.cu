#include "hip/hip_runtime.h"
#pragma  once
#include "hip/hip_runtime.h"
#include ""
#include <cv.h>
#include <highgui.h>
#include <stdio.h>
 #include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <Windows.h>
#include <time.h>

using namespace std;
using namespace cv;

	uchar *image;
	uchar *pcaface;
	uchar *mean;
	float *q;
	int test_km;

	uchar *gpu_image;
	uchar *gpu_pcaface;
	uchar *gpu_mean;
	float *gpu_distance;
	float *gpu_Q;
	dim3   grid1(256, 1);
	dim3   block1(256, 1);

__global__ void Edge(uchar * inMAP, uchar * outMAP) {

    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    int Dim = gridDim.x *blockDim.x;
    int offset = x + y *Dim;

}

__global__ void MatrixMul(uchar *M, float *N, float *P, int num_pca) 
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
	int offset = x;
	__shared__ uchar M_temp[256];
	__shared__ uchar N_temp[256];

	M_temp[threadIdx.x] = 0;
	N_temp[threadIdx.x] = 0;
	__syncthreads();
	for (int i = 0; i < num_pca; i++)
		M_temp[threadIdx.x] = M[i, offset];
	N_temp[threadIdx.x] = N[offset];
	__syncthreads();
	for(int i = 0; i < num_pca; i++)
		atomicAdd(&P[i], (float) M_temp[threadIdx.x] * N_temp[threadIdx.x]);
}

__global__ void MatrixTranverse(float *M)
{
	int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    int Dim = gridDim.x *blockDim.x;
    int offset = x + y *Dim; //global ID
	M[offset] = M[x * Dim + y];
}

__global__ void DCal(uchar *TestImage, uchar *AverageFace, float *distance) 
{
 //   int offset = threadIdx.x + blockDim.x*blockIdx.x;
/*	int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;
    int Dim = gridDim.x *blockDim.x;
    int offset = x + y *Dim;
	distance[ offset ] =  (float) (TestImage[ offset ] - AverageFace[ offset ]); */
}

extern "C" int run_initialization( Mat Image, Mat PcaFace, Mat Mean)
{
	uchar *image = (uchar *) Image.data;
	uchar *pcaface = (uchar *) PcaFace.data;
	uchar *mean = (uchar *) Mean.data;
	int PcaLength = PcaFace.rows;
	int PcaWidth = PcaFace.cols;
	size_t pitch;
	size_t host_orig_pitch = PcaWidth * sizeof(uchar);
	/**************************** start GPU initilzations **************************/
	hipError_t cudaStatus;    
	cudaStatus = hipSetDevice(0);
	assert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&gpu_mean, Image.rows * Image.cols * sizeof(uchar));
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMemcpy(gpu_mean, mean, Mean.rows * Mean.cols * sizeof(uchar), hipMemcpyHostToDevice);
	assert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&gpu_Q, 1 * PcaFace.rows * sizeof(float));
	assert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&gpu_distance, Image.rows * Image.cols * sizeof(char));
	assert(cudaStatus == hipSuccess);

	cudaStatus = hipMallocPitch(&gpu_pcaface, &pitch, PcaWidth * sizeof(uchar), PcaLength * sizeof(uchar));
    assert(cudaStatus == hipSuccess);

	hipMemcpy2D(gpu_pcaface, pitch, pcaface, host_orig_pitch, PcaWidth, PcaLength, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

	/*************************** CPU initializations ******************************/
	q = (float*) malloc(sizeof(float) * 1 * PcaFace.rows);
	return 0;
}

extern "C" int run_prepartion( Mat Image, Mat PcaFace, Mat Mean, float *Q)
{
	uchar *image = (uchar *) Image.data;
	uchar *pcaface = (uchar *) PcaFace.data;
	uchar *mean = (uchar *) Mean.data;
	uchar *image1;

	float *test_distance;
	hipError_t cudaStatus;
//	int test_km = 0;

	test_distance = (float *)malloc(sizeof(float)* Image.rows * Image.cols);
	cudaStatus = hipMalloc((void**)&gpu_image, Image.rows * Image.cols * sizeof(uchar));
	image1 = (uchar *)malloc(sizeof(uchar) * Image.rows * Image.cols);
	assert(cudaStatus == hipSuccess);

	cudaStatus = hipMemcpy(gpu_image, image, Image.rows * Image.cols * sizeof(uchar), hipMemcpyHostToDevice);
	assert(cudaStatus == hipSuccess);
	cudaStatus = hipMemcpy(image1, gpu_image, Image.rows * Image.cols * sizeof(uchar), hipMemcpyDeviceToHost);
	assert(cudaStatus == hipSuccess);

    /*********************** GPU Calculation: Calculating D *************************/
	DCal<<<grid1,block1>>>(gpu_image, gpu_mean, gpu_distance);
	cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
	cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMemcpy(test_distance, gpu_distance, Image.rows * Image.cols * sizeof(uchar), hipMemcpyDeviceToHost);
	assert(cudaStatus == hipSuccess);
	/*********************** GPU Calculation: Calculating Q**************************/
	MatrixMul<<<grid1,block1>>>(gpu_pcaface, gpu_distance, gpu_Q, PcaFace.rows);
	cudaStatus = hipMemcpy(q, gpu_Q, 1 * PcaFace.rows * sizeof(float), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

	for (int j=0; j<PcaFace.rows; j++)
		{
			Q[test_km + j] = q[j];
		}
		test_km += PcaFace.rows;
		return 0;
}

extern "C" int Cal_test_pic_Q( Mat Image, Mat PcaFace, Mat Mean, float *q_test)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(gpu_image, image, Image.rows * Image.cols * sizeof(uchar), hipMemcpyHostToDevice);
	assert(cudaStatus == hipSuccess);
    /*********************** GPU Calculation: Calculating D *************************/
	DCal<<<grid1,block1>>>(gpu_image, gpu_mean, gpu_distance);
	cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

	/*********************** GPU Calculation: Calculating Q**************************/
	MatrixMul<<<grid1,block1>>>(gpu_pcaface, gpu_distance, gpu_Q, PcaFace.cols);
	cudaStatus = hipMemcpy(q_test, gpu_Q, Image.rows * PcaFace.rows * sizeof(float), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);
	return 0;
}

extern "C" int deinitialization(void)
{
	hipFree(gpu_image);
	hipFree(gpu_pcaface);
	hipFree(gpu_mean);
	hipFree(gpu_distance);
	hipFree(gpu_Q);
	free(q);
	return 0;
}

extern "C" void run(IplImage *frame, Mat eigenVector, Mat eigenValue, Mat mean)
{
	uchar     *inMAP;
	uchar     *outMAP;
	uchar	  *distance;
	uchar	  *data = (uchar*)frame->imageData;
	uchar	  *eigen_vector = (uchar *) eigenVector.data;
	int		  H_eigenVector = eigenVector.rows;
	int		  W_eigenVector = eigenVector.cols;
	///////////////////////////////////////////
	hipError_t cudaStatus;    
	cudaStatus = hipSetDevice(0);
    assert(cudaStatus == hipSuccess);
	
	cudaStatus = hipMalloc((void**)&inMAP, frame->height*frame->width* sizeof(uchar));
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&outMAP, frame->height*frame->width* sizeof(uchar));
    assert(cudaStatus == hipSuccess);
	/////////////////////////////////////////////
	int size = frame->height* frame->width* sizeof(uchar);
    cudaStatus = hipMemcpy(inMAP, data, size, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

	dim3 grid(frame->width,frame->height);
    dim3 block(1,1);

	Edge<<<grid, block>>>(inMAP, outMAP);
	cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == hipSuccess);

	cudaStatus = hipMemcpy(data, outMAP, size, hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);
}